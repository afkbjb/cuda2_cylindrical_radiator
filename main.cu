#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include "radiator_cpu.h"     // CPU initialize/propagate/avg
#include "radiator_gpu.h"

static void check(hipError_t e){
  if(e!=hipSuccess){
    std::cerr<<"CUDA error: "<<hipGetErrorString(e)<<"\n";
    std::exit(1);
  }
}

static const int BX = 16;
static const int BY = 16;

int main(int argc,char**argv){
  int n=32,m=32,p=10;
  bool do_avg=false, skip_cpu=false, timing=false;

  // 解析
  for(int i=1;i<argc;i++){
    if(!strcmp(argv[i],"-n")&&i+1<argc) n=atoi(argv[++i]);
    else if(!strcmp(argv[i],"-m")&&i+1<argc) m=atoi(argv[++i]);
    else if(!strcmp(argv[i],"-p")&&i+1<argc) p=atoi(argv[++i]);
    else if(!strcmp(argv[i],"-a")) do_avg=true;
    else if(!strcmp(argv[i],"-c")) skip_cpu=true;
    else if(!strcmp(argv[i],"-t")) timing=true;
    else { std::cerr<<"Unknown "<<argv[i]<<"\n"; return 1; }
  }

  // 检查整除
  if(m%BX||n%BY){
    std::cerr<<"Error: "<<BX<<"x"<<BY<<" block must divide "<<n<<"x"<<m<<"\n";
    return 1;
  }
  dim3 block(BX,BY), grid(m/BX,n/BY);

  float *h_mat=nullptr,*h_next=nullptr;
  double cpu_t=0;
  if(!skip_cpu){
    h_mat  = new float[n*m];
    h_next = new float[n*m];
    initialize(h_mat,h_next,n,m);
    auto t0=std::chrono::high_resolution_clock::now();
    for(int it=0;it<p;it++){
      propagate_step(h_mat,h_next,n,m);
      std::swap(h_mat,h_next);
    }
    auto t1=std::chrono::high_resolution_clock::now();
    cpu_t = std::chrono::duration<double, std::milli>(t1-t0).count();
    if(timing) std::cout<<"CPU compute: "<<cpu_t<<" ms\n";
  }

  // GPU malloc
  size_t Mbytes = n*m*sizeof(float), Abytes=n*sizeof(float);
  float *d1,*d2,*dav;
  hipEvent_t e0,e1; check(hipEventCreate(&e0)); check(hipEventCreate(&e1));
  hipEventRecord(e0);
  check(hipMalloc(&d1,Mbytes));
  check(hipMalloc(&d2,Mbytes));
  check(hipMalloc(&dav,Abytes));
  hipEventRecord(e1); hipEventSynchronize(e1);
  float tAlloc; hipEventElapsedTime(&tAlloc,e0,e1);
  if(timing) std::cout<<"GPU alloc: "<<tAlloc<<" ms\n";

  // 如果 skip_cpu，自行初始化 h_mat/h_next
  if(skip_cpu){
    h_mat  = new float[n*m];
    h_next = new float[n*m];
    initialize(h_mat,h_next,n,m);
  }

  // H2D
  hipEventRecord(e0);
  check(hipMemcpy(d1,h_mat,Mbytes,hipMemcpyHostToDevice));
  check(hipMemcpy(d2,h_next,Mbytes,hipMemcpyHostToDevice));
  hipEventRecord(e1); hipEventSynchronize(e1);
  float tH2D; hipEventElapsedTime(&tH2D,e0,e1);
  if(timing) std::cout<<"GPU H2D: "<<tH2D<<" ms\n";

  // GPU propagate p 步
  hipEventRecord(e0);
  for(int it=0;it<p;it++){
    propagate_kernel<<<grid,block>>>(d1,d2,n,m);
    hipDeviceSynchronize();
    std::swap(d1,d2);
  }
  hipEventRecord(e1); hipEventSynchronize(e1);
  float tProp; hipEventElapsedTime(&tProp,e0,e1);
  if(timing) std::cout<<"GPU propagate: "<<tProp<<" ms\n";

  // GPU average
  hipEventRecord(e0);
  average_kernel<<<n,1>>>(d1,dav,n,m);
  hipDeviceSynchronize();
  hipEventRecord(e1); hipEventSynchronize(e1);
  float tAvg; hipEventElapsedTime(&tAvg,e0,e1);
  if(timing) std::cout<<"GPU average: "<<tAvg<<" ms\n";

  // D2H
  float *h_res=new float[n*m], *h_avr=new float[n];
  hipEventRecord(e0);
  check(hipMemcpy(h_res,d1,Mbytes,hipMemcpyDeviceToHost));
  check(hipMemcpy(h_avr,dav,Abytes,hipMemcpyDeviceToHost));
  hipEventRecord(e1); hipEventSynchronize(e1);
  float tDtoH; hipEventElapsedTime(&tDtoH,e0,e1);
  if(timing) std::cout<<"GPU D2H: "<<tDtoH<<" ms\n";

  // 比对
  if(!skip_cpu){
    int cntM=0,cntA=0; float maxM=0,maxA=0;
    for(int i=0;i<n*m;i++){
      float d=fabs(h_mat[i]-h_res[i]);
      if(d>1e-4) cntM++;
      maxM=fmax(maxM,d);
    }
    float *h_avc=new float[n];
    compute_averages(h_mat,h_avc,n,m);
    for(int i=0;i<n;i++){
      float d=fabs(h_avc[i]-h_avr[i]);
      if(d>1e-4) cntA++;
      maxA=fmax(maxA,d);
    }
    std::cout<<"Matrix mismatches: "<<cntM<<" max="<<maxM<<"\n";
    std::cout<<"Avg mismatches:    "<<cntA<<" max="<<maxA<<"\n";
    if(timing){
      double gpuComp = tProp + tAvg;
      std::cout<<"Speedup: "<<(cpu_t/gpuComp)<<"\n";
    }
    delete[] h_avc;
  }

  // -a 时打印行平均
  if(do_avg){
    std::cout<<"Row avgs (GPU):\n";
    for(int i=0;i<n;i++)
      std::cout<<"  "<<i<<": "<<h_avr[i]<<"\n";
  }

  // 释放
  delete[] h_mat; delete[] h_next;
  delete[] h_res; delete[] h_avr;
  hipFree(d1); hipFree(d2); hipFree(dav);
  hipEventDestroy(e0); hipEventDestroy(e1);
  return 0;
}
